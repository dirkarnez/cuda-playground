#include <iostream>
#include <cstdio>
#include <hip/hip_runtime.h>


using namespace std;

__global__ void vec_add(int *a, int *b)
{
    int i = blockIdx.x;
    a[i] += b[i];
}

int main()
{
    fprintf(stdout, "Hello world\n");
    
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess) {
        printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    } else {
        printf("Found %d GPUs\n");
    }
    
    const int N = 100;
    // ---- allocate the memory ---- //
    int *a, *b;
    hipMallocManaged(&a, N*sizeof(int));
    hipMallocManaged(&b, N*sizeof(int));
    // ---- initialize a,b ---- //
    for(int i=0;i<N;i++)
    {
        a[i] = i;
        b[i] = 2*i;
    }
    // ---- add b to a ---- //
    vec_add<<<N,1>>>(a,b);
    hipDeviceSynchronize();
    // ---- display a ---- //
    for(int i=0;i<N;i++)
    {
        cout << a[i] << endl;
    }
    
    return 0;
}
